#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

void cuda_gemm_128_128(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C);
void cuda_gemm_64_128(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C);
void cuda_gemm_64_64(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C);
void cuda_gemm_32_64(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C);

void cuda_gemm(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    if (N >4096 || M > 4096) {
        cuda_gemm_128_128(M, N, K, alpha, A, B, beta, C);
    }
    else if (N > 2048 || M>2048) {
        cuda_gemm_64_128(M, N, K, alpha, A, B, beta, C);
    }
    else if (N==1024 && M==1024) {
        cuda_gemm_32_64(M, N, K, alpha, A, B, beta, C);
    }
    else {
        cuda_gemm_64_64(M, N, K, alpha, A, B, beta, C);
    }
}